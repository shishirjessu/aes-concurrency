#include "hip/hip_runtime.h"
#include "aes-ctr.cuh"
#include "aes_seq.h"

#define MAX_THREADS 1024

void err_chk() {
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
	// else
	// 	printf("%s\n", "hipSuccess");
}

/* print out 16-byte block as grid */
__device__ void printBlock(unsigned char* state) {
	for (int x = 0; x < 4; x++) {
		for (int y = 0; y < 4; y++)
			printf("%x ", state[y * 4 + x]);

		printf("\n");
	}
}

/* left-shifts a row, data is is column order */
__device__ void leftRotateByOne(unsigned char* state, int row, int size) {
	char temp = state[row];
	int x;
	for (x = 0; x < size - 1; x++) {
		int cur = row;
		int next = row + 4;

		state[cur] = state[next];
		row += 4;
	}
	state[row] = temp;
}

__device__ unsigned char gmul (unsigned char a, unsigned char b) {
  if (b == 1) return a;
  if (b == 2) return d_gal2[a];
  if (b == 3) return d_gal3[a];

  return 0;
}


__device__ void mixSingleColumn(unsigned char* col) {
	unsigned char temp[4];

	for (int x = 0; x < 4; x++)
		temp[x] = col[x];

	col[0] = gmul(temp[0], 2) ^ gmul(temp[3], 1) ^ gmul(temp[2], 1) ^ gmul(temp[1], 3);
	col[1] = gmul(temp[1], 2) ^ gmul(temp[0], 1) ^ gmul(temp[3], 1) ^ gmul(temp[2], 3);
	col[2] = gmul(temp[2], 2) ^ gmul(temp[1], 1) ^ gmul(temp[0], 1) ^ gmul(temp[3], 3);
	col[3] = gmul(temp[3], 2) ^ gmul(temp[2], 1) ^ gmul(temp[1], 1) ^ gmul(temp[0], 3);
}

/* the main steps of AES */

/* prepares one 4-byte word */
void keyScheduleCore (unsigned char* word, int round) {
	char temp = word[0];
	word[0] = word[1];
	word[1] = word[2];
	word[2] = word[3];
	word[3] = temp;

	for (int x = 0; x < 4; x++)
		word[x] = sub_bytes_lookup[word[x]];

	word[0] ^= rcon[round];
}


/* expand them keys brah
   n: number of bytes in the original key
   b: number of total bytes we want
*/
void keyExpansion (unsigned char* key, unsigned char* expandedKeys, int n, int b) {
	int numExp;
	for (numExp = 0; numExp < n; numExp++)
		expandedKeys[numExp] = key[numExp];

	int round = 1;
	while (numExp < b) {
		/* copy over the last 4 bytes of the expanded key to temp */
		unsigned char temp[4] = {0, 0, 0, 0};
		//unsigned char* temp = (unsigned char*) calloc(1, 4);
		for (int x = numExp - 4, y = 0; y < 4; x++, y++)
			temp[y] = expandedKeys[x];

		/* perform the core on temp, incrementing round when done */
		keyScheduleCore(temp, round++);

		/* x-or temp with whatever's n bytes before it, then drop it in expandedKeys*/
		for (int x = numExp - n, y = 0; y < 4; y++, x++)
			temp[y] ^= expandedKeys[x];

		for (int x = 0; x < 4; x++)
			expandedKeys[numExp + x] = temp[x];

		/* now we've expanded 4 more bytes */
		numExp += 4;

		/* need to produce the next 12 bytes of expanded key */
		for (int a = 0; a < 3; a++) {
			/* grab the previous 4 bytes and put it in temp */
			for (int x = numExp - 4, y = 0; y < 4; x++, y++)
				temp[y] = expandedKeys[x];

			/* x-or temp with whatever's n bytes before it, then drop it in expandedKeys*/
			for (int x = numExp - n, y = 0; y < 4; y++, x++)
				temp[y] ^= expandedKeys[x];

			for (int x = 0; x < 4; x++)
				expandedKeys[numExp + x] = temp[x];

			numExp += 4;
		}
	}
}

/* switch data with corresponding data in Rijndael s-box */
__device__ void subBytes (unsigned char* state) {
	for (int x = 0; x < 16; x++)
		state[x] = d_sub_bytes_lookup[state[x]];
}

__device__ void addRoundKey (unsigned char* state, unsigned char* key) {
	for (int x = 0; x < 16; x++)
		state[x] ^= key[x];
}

__device__ void xOr (unsigned char* a, unsigned char* b) {
	addRoundKey(a, b);
}

__device__ void shiftRows (unsigned char* state) {
	for (int x = 1; x <= 3; x++) {
		for (int y = 0; y < x; y++)
			leftRotateByOne(state, x, 4);
	}
}

/* perform the mix columns operation
   n: the number of columns */
__device__ void mixColumns (unsigned char* state, int n) {
	for (int x = 0; x < n; x++) {
		unsigned char col [4] = {0, 0, 0, 0};
		//unsigned char* col = (unsigned char*) calloc(1, 4);
		for (int y = x * 4, z = 0; z < 4; y ++, z++) {
			col[z] = state[y];
		}

		mixSingleColumn(col);

		for (int y = x * 4, z = 0; z < 4; y ++, z++)
			state[y] = col[z];

    	free(col);
	}
}

__global__ void encrypt(unsigned char* state, unsigned char* expandedKeys, int bufferSize) {
	long cVal = blockIdx.x*blockDim.x+threadIdx.x;
	__syncthreads();
	// if (cVal == 0) {
	// 	for (int x = 0; x < bufferSize; x++)
 //  			printf("%x\n", state[x]);
	// }
	__syncthreads();

	long nonce = 0xaaaaaaaaaaaaaaaa;

	long counter [2];

	counter[0] = cVal;
	counter[1] = nonce;

	unsigned char* counterState = (unsigned char*) counter;

	addRoundKey(counterState, expandedKeys);

	for (int x = 1; x < 11; x++) {
		subBytes(counterState);
		shiftRows(counterState);

		if (x != 10)
			mixColumns(counterState, 4);

		addRoundKey(counterState, expandedKeys + 16 * x);
	}

	int blockSize = 16;
	unsigned char* toXor = state + (blockSize * cVal);

	xOr(toXor, counterState);

	__syncthreads();
	// if (cVal == 0) {
	// 	for (int x = 0; x < bufferSize; x++)
 //  			printf("%d %x\n", x, state[x]);
	// }
}

void runAES(unsigned char* state, int stateLength, int blockSize, unsigned char* key) {
	/* to get correct size for padded buffer */
	int numBlocks = (stateLength / blockSize) + (stateLength % blockSize != 0);
	int bufferSize = numBlocks * blockSize;

	unsigned char* result = (unsigned char*) calloc(1,  bufferSize);

	int x;
	for (x = 0; x < stateLength; x++)
		result[x] = state[x];

	/* we need even block sizes, so we will pad any uneven block */
	char diff = blockSize - (stateLength % blockSize);
	while (x < bufferSize) {
		result[x] = diff;
		x++;
	}

	int numExpandedKeyBytes = 176;

	unsigned char* expandedKeys = (unsigned char*) calloc(1, numExpandedKeyBytes);

	/* Key expansion - once */
  	keyExpansion(key, expandedKeys, 16, numExpandedKeyBytes);

  	/* Throw stuff onto GPU */
  	unsigned char *dState, *dExpandedKeys; /* device ptrs */
  	hipMalloc((void**)&dState, bufferSize);
  	err_chk();

  	hipMemcpy(dState, result, bufferSize, hipMemcpyHostToDevice);
  	err_chk();

  	hipMalloc((void**)&dExpandedKeys, numExpandedKeyBytes);
  	err_chk();

  	hipMemcpy(dExpandedKeys, expandedKeys, numExpandedKeyBytes, hipMemcpyHostToDevice);
  	err_chk();

  	int numThreadBlocks = (numBlocks + MAX_THREADS - 1) / MAX_THREADS;
  	int threadsPerBlock = numBlocks / numThreadBlocks;

  	// printf("num threads: %d %d %d\n", numThreadBlocks, threadsPerBlock, numThreadBlocks * threadsPerBlock);

  	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

  	encrypt <<<numThreadBlocks, threadsPerBlock>>> (dState, dExpandedKeys, bufferSize);
  	err_chk();

  	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n", milliseconds/1000.0);
}

char* read_input(char* input) {
	/* Open file and read in first 2 things*/
	FILE* input_file = fopen(input, "r");
	char* result;
	if (input_file != NULL) {
		fseek(input_file, 0L, SEEK_END);
		long s = ftell(input_file);
		rewind(input_file);

		result = (char*) malloc(s);
		if (result != NULL) {
			fread(result, s, 1, input_file);
			fclose(input_file);
			input_file = NULL;
		}
	}

	if (input_file != NULL) fclose(input_file);

	int len = strlen(result);
	if( result[len-1] == '\n' )
	    result[len-1] = 0;

	return result;
}

int main (int argc, char** argv) {
	char* keyFile = argv[1];
	char* stateFile = argv[2];

	char* key = read_input(keyFile);
	char* state = read_input(stateFile);

  	runAES((unsigned char*) state, strlen(state), 16, (unsigned char*) key);

	free(state);
  	free(key);
}
